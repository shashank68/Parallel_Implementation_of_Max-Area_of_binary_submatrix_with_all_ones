
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

__global__
void MaxSizeSubMat(int* d_M, int* d_S, int* d_c, int* d_r, int* N) {
	int k = threadIdx.x, i , j;
	if(*N >= (*d_c + *d_r)/2) {
		i = *d_r - (k + 1);
		j = *N + k + 2 - *d_r;
	} else {
		i = *N - k;
		j = k + 1;
	}
	if(j >= *d_c || j <= 0 || i <= 0 || i >= *d_r) {
		return;
	}
	if(d_M[ (*d_c * i) + j ] == 1) {
		int a = d_S[ (*d_c * i) + j-1] , b = d_S[ (*d_c * (i-1)) + j ], c = d_S[ (*d_c * (i-1)) + j -1 ];
		if(a <= b && a <= c) {
			d_S[ (*d_c * i) + j ] = a + 1;
			__syncthreads();
		} else if(b <= c && b <= a) {
			d_S[ (*d_c * i) + j ] = b + 1;
			__syncthreads();
		}else {
			d_S[ (*d_c * i) + j ] = c + 1;
			__syncthreads();
		}
	} else {
		d_S[ (*d_c * i) + j ] = 0;
	}
}



int main() {
	int isize = sizeof(int) ; //bsize = sizeof(bool);
	int h_r, h_c;
	//int *d_r, *d_c;
	cout << "Enter the Number of rows in the matrix." << endl;
	cin >> h_r;
	cout << "Enter the Number of columns in the matrix." << endl;
	cin >> h_c;

	int h_M[h_r][h_c];

	int  h_S[h_r][h_c];
	cout << "Enter the elements of the matrix.\n";
	for(int i = 0; i < h_r; i++) {
		for(int j = 0; j < h_c; j++) {
			cin >> h_M[i][j];
			if(i == 0 || j == 0) {
				h_S[i][j] = h_M[i][j];
			} else {

				h_S[i][j] = 0;
			}
		}
	}
	int *d_M, *d_c, *d_S, *d_r;

	hipMalloc((void**)&d_M, h_r * h_c * isize);
	hipMalloc((void**)&d_S, h_r * h_c * isize);
	hipMalloc((void**)&d_c, isize);
	hipMalloc((void**)&d_r, isize);

	for(int i = 0; i < h_r; i++) {
		for(int j = 0; j < h_c; j++) {
			hipMemcpy( &d_M[ (h_c * i) + j ], &h_M[i][j], isize, hipMemcpyHostToDevice);
			hipMemcpy( &d_S[ (h_c * i) + j ], &h_S[i][j], isize, hipMemcpyHostToDevice);
		}
	}
	puts("");
	hipMemcpy(d_c, &h_c, isize, hipMemcpyHostToDevice);
	hipMemcpy(d_r, &h_r, isize, hipMemcpyHostToDevice);

	int n = (h_r + h_c - 3), *N;
	hipMalloc((void**)&N, isize);

	for(int i = 1; i <= (n + 1)/2; i++) {
		hipMemcpy( N, &i, isize, hipMemcpyHostToDevice);
		MaxSizeSubMat<<<1,i>>> (d_M, d_S, d_c, d_r, N);
		hipDeviceSynchronize();
	}
	for(int i = (n + 3)/2; i <= n ; i++ ) {
		hipMemcpy( N, &i, isize, hipMemcpyHostToDevice);
		MaxSizeSubMat<<<1, (n - i + 1)>>> (d_M, d_S, d_c, d_r, N);
		hipDeviceSynchronize();
	}
	for(int i = 0; i < h_r; i++) {
		for(int j = 0; j < h_c; j++) {
			hipMemcpy( &h_S[i][j], &d_S[ (h_c * i) + j ], isize, hipMemcpyDeviceToHost);
		}
	}
	hipDeviceSynchronize();
	int max = h_S[0][0];
	//cout << " The square size matrix is" << endl;
	for(int i = 0; i < h_r; i++) {
		for(int j = 0; j < h_c; j++) {
			if(max < h_S[i][j]) {
				max = h_S[i][j];
			}
			//cout << h_S[i][j] << " ";
		}
		//cout << endl;
	}
	cout << "Max area is " << max * max << ".\n";
	hipFree(d_M); hipFree(d_S);
}

